#include "hip/hip_runtime.h"
/**
 * //TODO:
 * General:
 * - need to split this into cpp, h, cu
 * - cpp and h compiled with g++, and nvcc for cu files
 * - how to separate compilation in cmake and then later link?
 * - implement same function in cpu, then compare
 * - see if Thrust library can help
 * 
 * Specific:
 * - using unsigned int for histogram data yields 1MB of data, which most are useless
 * - atomicAdd accepts only 32/64-bit word (int, float, unsigned int, etc)
 * - possible solution atomicAdd for short (16-bit) --> https://devtalk.nvidia.com/default/topic/495219/cuda-programming-and-performance/how-to-use-atomiccas-to-implement-atomicadd-short-trouble-adapting-programming-guide-example/
 * 
 * Needed features:
 * - mean
 * - min
 * - max
 * - variance
 * - kurtosis
 * - skewness
 * - central moment 1st to 5th order
 * 
 */
 
#include <cstdio>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>

#include "kernCalcCornerBlockHist.cu"
#include "kernCalcBlockHist.cu"
#include "kernCalcStatistics.cu"

using namespace std;
using namespace cv;

void cpuCalcBlockHist(const Mat src, int blockSizeX, int blockSizeY, int beginX, int beginY, unsigned int * outHist){
	unsigned char *input = (unsigned char*) src.data;
	int count = 0;
	int bin;
	for (int j = beginY; j < beginY + blockSizeY; j++){
		for (int i = beginX; i < beginX + blockSizeX; i++){
			bin = input[src.rows * j + i];
			//cout<<"index = "<<src.rows * j + i<<endl;
			outHist[bin]++;
			count++;
		}
	}
	cout<<endl;
	cout<<"count = "<<count<<endl;
}

void processHistogram(unsigned int * hist, int max, bool show=false){
	float mean = 0;
	float sum = 0;
	int n = 0;
	
	for (int i = 0; i < max; i++){
		if (show)
			printf("%d\t%d\n", i, hist[i]);
		sum += i * hist[i];
		n += hist[i];
	}
	mean = sum / (float)n;
	printf("sum = %f\n", sum);
	printf("n = %d\n", n);
	printf("mean = %.5f\n", mean);
}

void processPseudoHistogram(unsigned int * hist, int dimx, int dimy, int dimz, int pitch, int blockX, int blockY, bool show=false){
	float mean = 0;
	float sum = 0;
	int n = 0;

	int idx;
	for (int i = 0; i < dimz; i++){
		idx = pitch * (dimx * blockY + blockX) + i;
		if (show)
			printf("%d\t%d\n", i, hist[idx]);
		sum += i * hist[idx];
		n += hist[idx];
	}
	
	/**
	// loop through all elements is not necessary
	for (int k = blockX; k < dimx; k++){
		for (int j = blockY; j < dimy; j++){
			for (int i = 0; i < dimz; i++){
				idx = pitch * (dimx * j + k) + i;
				if (show)
					printf("%d\t%d\n", i, hist[idx]);
				sum += i * hist[idx];
				n += hist[i];
			}
		}
	}
	*/
	
	mean = sum / (float)n;
	printf("sum = %f\n", sum);
	printf("n = %d\n", n);
	printf("mean = %.5f\n", mean);
}

int main (int argc, char** argv){
	Mat matSrc;
	
	if (argc == 2){
		matSrc = imread(argv[1], 0);
	} else {
		printf("Usage: %s [image file]!\n", argv[0]);
		return -1;
	}
	
	// which block to show result for testing purpose
	int tmp_whichBlockX = 30;
	int tmp_whichBlockY = 30; // referring to gpu block 0-30
	
	// block sizes
	int imgBlockSizeX = 32, imgBlockSizeY = 32;
	//int beginX = 480, beginY = 480;
	int beginX = tmp_whichBlockX * 16;
	int beginY = tmp_whichBlockY * 16;

	// cuda grid and thread
	dim3 blocksPerGrid;
	dim3 threadsPerBlock;
	
	// cuda timers
	hipEvent_t start, stop;
	float time_kernel;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	// image data
	unsigned char * host_image = matSrc.data;
	unsigned int host_hist[256] = {0};
	
	// device image
	unsigned char * dev_image;
	size_t size = matSrc.rows * matSrc.cols * sizeof(unsigned char);
		
	// stride for block processing overlap
	int strideX = 16, strideY = 16;
	
	// grids and thread for cuda
	int gpuBlockTotalX = matSrc.cols / strideX;
	int gpuBlockTotalY = matSrc.rows / strideY;
	blocksPerGrid = dim3(gpuBlockTotalX-1, gpuBlockTotalY-1, 1);
	threadsPerBlock = dim3(imgBlockSizeX, imgBlockSizeY, 1);
	
	// histogram, pseudo multi-dimension array
	unsigned int host_hist2[gpuBlockTotalX*gpuBlockTotalY*256];
	unsigned int * dev_hist2;
	int dev_hist2_pitch = 256;
	size_t size_hist2 = gpuBlockTotalX * gpuBlockTotalY * 256 * sizeof(unsigned int);
	
	// main show
	printf("=============\n");
	printf("Running the real deal\n");
	printf("blocks per grid = (%d, %d)\n", gpuBlockTotalX-1, gpuBlockTotalY-1);
	printf("threads per block = (%d, %d)\n", imgBlockSizeX, imgBlockSizeY);
	
	// timer begin
	hipEventRecord(start,0);
	
	// allocating and copying memory in device
	hipMalloc(&dev_image, size);
	hipMemcpy(dev_image, host_image, size, hipMemcpyHostToDevice);	
	hipMalloc(&dev_hist2, size_hist2);
	hipMemset(dev_hist2, 0, size_hist2);
	
	// kernel call
	kernCalcBlockHist<<<blocksPerGrid, threadsPerBlock>>>(dev_image, matSrc.rows, matSrc.cols, strideX, strideY, dev_hist2, dev_hist2_pitch);

	// copy the result back
	hipMemcpy(host_hist2, dev_hist2, size_hist2, hipMemcpyDeviceToHost);
	
	// timer end
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	
	// print out time
	hipEventElapsedTime(&time_kernel, start, stop);
	printf("Whole image GPU histogram took %.5f ms\n", time_kernel);

	// testing result
	printf("\nhistogram for block (%d,%d) from real deal\n", tmp_whichBlockX, tmp_whichBlockY);
	processPseudoHistogram(host_hist2, gpuBlockTotalX, gpuBlockTotalY, dev_hist2_pitch, 256, tmp_whichBlockX, tmp_whichBlockY, false);
	
	
	// ================================ reference block calculation =================================
	
	blocksPerGrid = dim3(1,1,1);
	threadsPerBlock = dim3(imgBlockSizeX, imgBlockSizeY, 1);
	
	printf("\n\n===========\n");
	printf("reference calculation\n");
	
	// corner histogram
	hipEventRecord(start, 0);
	cpuCalcBlockHist(matSrc, imgBlockSizeX, imgBlockSizeY, beginX, beginY, host_hist);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	
	hipEventElapsedTime(&time_kernel, start, stop);
	printf("CPU Histogram took %.5f ms\n", time_kernel);
	
	printf("Histogram from CPU, result from block (%d,%d)\n", tmp_whichBlockX, tmp_whichBlockY);
	processHistogram(host_hist, 256);
	
	/**
	// testing cuprintf
	printf("testing cuprintf\n");
	cudaPrintfInit();
	//kernCalcBlockHist<<<dim3(2,2), dim3(2,2)>>>();
	cudaPrintfDisplay(stdout, true);
	cudaPrintfEnd();
	*/
	
	// cleanup
	hipFree(dev_image);
//	hipFree(dev_hist);
	hipFree(dev_hist2);
	hipDeviceReset();
	return 0;
}
