#include "hip/hip_runtime.h"
/**
 * //TODO:
 * - need to split this into cpp, h, cu
 * - cpp and h compiled with g++, and nvcc for cu files
 * - how to separate compilation in cmake and then later link?
 * 
 */
#include <cstdio>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>

#include "kernCalcCornerBlockHist.cu"
#include "kernCalcBlockHist.cu"

using namespace std;
using namespace cv;

void processHistogram(unsigned int * hist, int max, bool show=false){
	float mean = 0;
	float sum = 0;
	int n = 0;
	
	for (int i = 0; i < max; i++){
		if (show)
			printf("%d\t%d\n", i, hist[i]);
		sum += i * hist[i];
		n += hist[i];
	}
	mean = sum / (float)n;
	printf("sum = %f\n", sum);
	printf("n = %d\n", n);
	printf("mean = %.5f\n", mean);
}

int main (int argc, char** argv){
	Mat matSrc;
	
	if (argc == 2){
		matSrc = imread(argv[1], 0);
	} else {
		printf("define input file!\n");
		return -1;
	}
	
	// block sizes
	int blockSizeX = 32, blockSizeY = 32;
	int beginX = 0, beginY = 0;
	
	// image data
	unsigned char * host_image = matSrc.data;
	unsigned int host_hist[256] = {0};
	
	// device image
	unsigned char * dev_image;
	size_t size = matSrc.rows * matSrc.cols * sizeof(unsigned char);
	hipMalloc(&dev_image, size);
	hipMemcpy(dev_image, host_image, size, hipMemcpyHostToDevice);
	
	// device histogram
	unsigned int * dev_hist;
	hipMalloc(&dev_hist, 256 * sizeof(unsigned int));
	hipMemset(dev_hist, 0, 256 * sizeof(unsigned int));
	
	// cuda grid and thread
	dim3 blocksPerGrid = dim3(1,1,1);
	dim3 threadsPerBlock = dim3(blockSizeX, blockSizeY, 1);
	
	// cuda timers
	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	// main show
	hipEventRecord(start, 0);
	kernCalcCornerBlockHist<<<blocksPerGrid, threadsPerBlock>>>(dev_image, matSrc.rows, matSrc.cols, beginX, beginY, dev_hist);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("GPU Histogram took %.5f ms\n", elapsedTime);
	
	hipMemcpy(host_hist, dev_hist, 256 * sizeof(unsigned int), hipMemcpyDeviceToHost);
	
	// printing result
	printf("Histogram from GPU\n");
	processHistogram(host_hist, 256);
	
	// testing cuprintf
	printf("testing cuprintf\n");
	cudaPrintfInit();
	kernCalcBlockHist<<<dim3(2,2), dim3(2,2)>>>();
	cudaPrintfDisplay(stdout, true);
	cudaPrintfEnd();
	
	// cleanup
	hipFree(dev_image);
	hipFree(dev_hist);
	hipDeviceReset();
	return 0;
}
