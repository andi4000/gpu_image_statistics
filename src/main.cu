#include "hip/hip_runtime.h"
/**
 * //TODO:
 * General:
 * - need to split this into cpp, h, cu
 * - cpp and h compiled with g++, and nvcc for cu files
 * - how to separate compilation in cmake and then later link?
 * - implement same function in cpu, then compare
 * - see if Thrust library can help 
 * - https://github.com/thrust/thrust/blob/master/examples/summary_statistics.cu
 * 
 * Specific:
 * - using unsigned int for histogram data yields 1MB of data, which most are useless
 * - atomicAdd accepts only 32/64-bit word (int, float, unsigned int, etc)
 * - possible solution atomicAdd for short (16-bit) --> https://devtalk.nvidia.com/default/topic/495219/cuda-programming-and-performance/how-to-use-atomiccas-to-implement-atomicadd-short-trouble-adapting-programming-guide-example/
 * 
 * Needed features:
 * - mean
 * - min
 * - max
 * - variance
 * - kurtosis
 * - skewness
 * - central moment 1st to 5th order
 * 
 */
 
#include "common.h"

#include "kernCalcCornerBlockHist.cu"
#include "kernCalcBlockHist.cu"
#include "kernCalcStatistics.cu"
#include "cpuCalculations.h"

using namespace std;
using namespace cv;

// Ref: http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char * file, int line, bool abort=true){
	if (code != hipSuccess){
		fprintf(stderr, "GPU Assert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

int main (int argc, char** argv){
	Mat matSrc;
	
	if (argc == 2){
		matSrc = imread(argv[1], 0);
	} else {
		printf("Usage: %s [image file]!\n", argv[0]);
		return -1;
	}
	
	// which block to show result for testing purpose
	int tmp_whichBlockX = 30;
	int tmp_whichBlockY = 30; // referring to gpu block 0-30
	
	// image block sizes
	int imgBlockSizeX = 32, imgBlockSizeY = 32;
		
	// stride for block processing overlap
	int strideX = 16, strideY = 16;
	
	// total blocks for cuda
	int gpuBlockTotalX = matSrc.cols / strideX - 1;
	int gpuBlockTotalY = matSrc.rows / strideY - 1;
	
	// cuda grid and thread
	dim3 blocksPerGrid;
	dim3 threadsPerBlock;
	
	// cuda timers
	hipEvent_t start, stop;
	float time_cpuHist, time_gpuHist, time_cpuStatCalc, time_gpuStatCalc;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	// image data
	unsigned char * host_image = matSrc.data;
	
	// device image
	unsigned char * dev_image;
	size_t size = matSrc.rows * matSrc.cols * sizeof(unsigned char);

	
	// ================================ CPU Histogram =================================
	
	int cpuHistPitch = 256;
	unsigned int cpuHist[gpuBlockTotalX * gpuBlockTotalY * cpuHistPitch];
	memset(cpuHist, 0, gpuBlockTotalX * gpuBlockTotalY * cpuHistPitch * sizeof(unsigned int));
		
	blocksPerGrid = dim3(1,1,1);
	threadsPerBlock = dim3(imgBlockSizeX, imgBlockSizeY, 1);
	
	printf("\n============= CPU =============\n");
	
	hipEventRecord(start, 0);
	cpuCalcHistAll(matSrc, imgBlockSizeX, imgBlockSizeY, gpuBlockTotalX, gpuBlockTotalY, strideX, strideY, cpuHist, cpuHistPitch);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	
	hipEventElapsedTime(&time_cpuHist, start, stop);
	printf("Whole image CPU Histogram took %.5f ms\n", time_cpuHist);
	
	printf("\nHistogram sample from CPU, block (%d,%d)\n", tmp_whichBlockX, tmp_whichBlockY);
	processPseudoHistogram(cpuHist, gpuBlockTotalX, gpuBlockTotalY, cpuHistPitch, 256, tmp_whichBlockX, tmp_whichBlockY, false);
	
	
	// =============================== CPU mean median max min  && central moments, skewness, kurtosis ====================
	
	// mean, median, max, min, serialized array
	int cpuStatPitch = gpuBlockTotalX; // 31
	float cpuStatMean[gpuBlockTotalX*gpuBlockTotalY];
	unsigned int cpuStatMedian[gpuBlockTotalX*gpuBlockTotalY];
	unsigned int cpuStatMax[gpuBlockTotalX*gpuBlockTotalY];
	unsigned int cpuStatMin[gpuBlockTotalX*gpuBlockTotalY];

	// variables to hold central moments
	float cpuStatCentralMoment2[gpuBlockTotalX*gpuBlockTotalY];
	float cpuStatCentralMoment3[gpuBlockTotalX*gpuBlockTotalY];
	float cpuStatCentralMoment4[gpuBlockTotalX*gpuBlockTotalY];
	float cpuStatCentralMoment5[gpuBlockTotalX*gpuBlockTotalY];
	memset(cpuStatCentralMoment2, 0.0, gpuBlockTotalX*gpuBlockTotalY*sizeof(float));
	memset(cpuStatCentralMoment3, 0.0, gpuBlockTotalX*gpuBlockTotalY*sizeof(float));
	memset(cpuStatCentralMoment4, 0.0, gpuBlockTotalX*gpuBlockTotalY*sizeof(float));
	memset(cpuStatCentralMoment5, 0.0, gpuBlockTotalX*gpuBlockTotalY*sizeof(float));

	// variance, skewness, and kurtosis
	float cpuStatVariance[gpuBlockTotalX*gpuBlockTotalY];
	float cpuStatSkewness[gpuBlockTotalX*gpuBlockTotalY];
	float cpuStatKurtosis[gpuBlockTotalX*gpuBlockTotalY];
	memset(cpuStatVariance, 0.0, gpuBlockTotalX*gpuBlockTotalY*sizeof(float));
	memset(cpuStatSkewness, 0.0, gpuBlockTotalX*gpuBlockTotalY*sizeof(float));
	memset(cpuStatKurtosis, 0.0, gpuBlockTotalX*gpuBlockTotalY*sizeof(float));

	// timer start
	hipEventRecord(start, 0);
	
	// calculating mean median max min
	cpuCalcMeanMedianMaxMin(cpuHist, gpuBlockTotalX, gpuBlockTotalY, cpuHistPitch, cpuStatMean, cpuStatMedian, cpuStatMax, cpuStatMin);

	// calculating moments
	cpuCalcCentralMoments(cpuHist, gpuBlockTotalX, gpuBlockTotalY, cpuHistPitch, cpuStatMean, cpuStatPitch, (imgBlockSizeX*imgBlockSizeY), cpuStatCentralMoment2, cpuStatCentralMoment3, cpuStatCentralMoment4, cpuStatCentralMoment5);
	
	// calculating variance, skewness and kurtosis
	cpuCalcVarianceSkewnessKurtosis(cpuStatCentralMoment2, cpuStatCentralMoment3, cpuStatCentralMoment4, gpuBlockTotalX, gpuBlockTotalY, cpuStatPitch, cpuStatVariance, cpuStatSkewness, cpuStatKurtosis);

	// timer stop & calculate elapsed time
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time_cpuStatCalc, start, stop);
	
	printf("\nCPU statistical calculation took %.5f ms\n", time_cpuStatCalc);
	printf(">>> SAMPLE for block (%d, %d)\n", tmp_whichBlockX, tmp_whichBlockY);
	printf("mean = %f\n", cpuStatMean[cpuStatPitch*tmp_whichBlockY + tmp_whichBlockX]);
	printf("median = %d\n", cpuStatMedian[cpuStatPitch*tmp_whichBlockY + tmp_whichBlockX]);
	printf("max = %d\n", cpuStatMax[cpuStatPitch*tmp_whichBlockY + tmp_whichBlockX]);
	printf("min = %d\n", cpuStatMin[cpuStatPitch*tmp_whichBlockY + tmp_whichBlockX]);
	
	printf("\nCentral Moments\n");
	printf("M2 = %.3f\n", cpuStatCentralMoment2[cpuStatPitch*tmp_whichBlockY + tmp_whichBlockX]);
	printf("M3 = %.3f\n", cpuStatCentralMoment3[cpuStatPitch*tmp_whichBlockY + tmp_whichBlockX]);
	printf("M4 = %.3f\n", cpuStatCentralMoment4[cpuStatPitch*tmp_whichBlockY + tmp_whichBlockX]);
	printf("M5 = %.3f\n", cpuStatCentralMoment5[cpuStatPitch*tmp_whichBlockY + tmp_whichBlockX]);
	
	printf("\n");
	printf("variance = %.3f\n", cpuStatVariance[cpuStatPitch*tmp_whichBlockY + tmp_whichBlockX]);
	printf("skewness = %.3f\n", cpuStatSkewness[cpuStatPitch*tmp_whichBlockY + tmp_whichBlockX]);
	printf("kurtosis = %.3f\n", cpuStatKurtosis[cpuStatPitch*tmp_whichBlockY + tmp_whichBlockX]);

	// ================================ GPU Histogram =================================
	
	// block sizes
	blocksPerGrid = dim3(gpuBlockTotalX, gpuBlockTotalY, 1);
	threadsPerBlock = dim3(imgBlockSizeX, imgBlockSizeY, 1);
	
	// histogram, pseudo multi-dimension array
	int dev_hist2_pitch = 256;
	unsigned int host_hist2[gpuBlockTotalX*gpuBlockTotalY*dev_hist2_pitch];
	unsigned int * dev_hist2;
	size_t size_hist2 = gpuBlockTotalX * gpuBlockTotalY * dev_hist2_pitch * sizeof(unsigned int);
	
	// main show
	printf("\n\n============= GPU =============\n");
	printf("blocks per grid = (%d, %d)\n", blocksPerGrid.x, blocksPerGrid.y);
	printf("threads per block = (%d, %d)\n", threadsPerBlock.x, threadsPerBlock.y);
	
	// timer begin
	hipEventRecord(start,0);
	
	// allocating and copying memory in device
	hipMalloc(&dev_image, size);
	hipMemcpy(dev_image, host_image, size, hipMemcpyHostToDevice);	
	hipMalloc(&dev_hist2, size_hist2);
	hipMemset(dev_hist2, 0, size_hist2);
	
	// kernel call
	kernCalcBlockHist<<<blocksPerGrid, threadsPerBlock>>>(dev_image, matSrc.rows, matSrc.cols, strideX, strideY, dev_hist2, dev_hist2_pitch);

	// copy the result back
	hipMemcpy(host_hist2, dev_hist2, size_hist2, hipMemcpyDeviceToHost);
	
	// timer end
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	
	// print out time
	hipEventElapsedTime(&time_gpuHist, start, stop);
	printf("Whole image GPU histogram took %.5f ms, %.2fx %s than CPU\n", time_gpuHist, (time_cpuHist/time_gpuHist), (time_gpuHist<time_cpuHist)?"faster":"slower");

	// testing result
	printf("\nHistogram sample from GPU, block (%d,%d)\n", tmp_whichBlockX, tmp_whichBlockY);
	//TODO: needed no more?
	processPseudoHistogram(host_hist2, gpuBlockTotalX, gpuBlockTotalY, dev_hist2_pitch, 256, tmp_whichBlockX, tmp_whichBlockY, false);

	// =============================== GPU mean median max min ====================
	// gpu histogram
	
	// out array size and pitch
	int statArraySize = gpuBlockTotalX*gpuBlockTotalY;
	int statArrayPitch = gpuBlockTotalX;
	
	// host variables: mean median max min
	float host_statMean[statArraySize];
	unsigned int host_statMedian[statArraySize];
	unsigned int host_statMax[statArraySize];
	unsigned int host_statMin[statArraySize];
	
	// host variables: central moments
	float host_statCentralMoment2[statArraySize];
	float host_statCentralMoment3[statArraySize];
	float host_statCentralMoment4[statArraySize];
	float host_statCentralMoment5[statArraySize];
	
	// host variables: variance, skewness and kurtosis
	float host_statVariance[statArraySize];
	float host_statSkewness[statArraySize];
	float host_statKurtosis[statArraySize];
	
	// device variables: mean median max min
	unsigned int * dev_hist2stat;
	float * dev_statMean;
	unsigned int * dev_statMedian;
	unsigned int * dev_statMax;
	unsigned int * dev_statMin;
	
	// device variables: central moments
	float * dev_statCentralMoments2;
	float * dev_statCentralMoments3;
	float * dev_statCentralMoments4;
	float * dev_statCentralMoments5;
	
	// device variables: skewness and kurtosis
	float * dev_statSkewness;
	float * dev_statKurtosis;
	
	// timer start
	hipEventRecord(start, 0);
	
	//TODO: why cant we use histogram data from last operation?
	// device malloc: histogram
	gpuErrChk( hipMalloc(&dev_hist2stat, size_hist2) );
	
	// device malloc: mean median max min
	gpuErrChk( hipMalloc(&dev_statMean, statArraySize * sizeof(float)) );
	gpuErrChk( hipMalloc(&dev_statMedian, statArraySize * sizeof(unsigned int)) );
	gpuErrChk( hipMalloc(&dev_statMax, statArraySize * sizeof(unsigned int)) );
	gpuErrChk( hipMalloc(&dev_statMin, statArraySize * sizeof(unsigned int)) );
	
	// device malloc: central moments
	gpuErrChk( hipMalloc(&dev_statCentralMoments2, statArraySize * sizeof(float)) );
	gpuErrChk( hipMalloc(&dev_statCentralMoments3, statArraySize * sizeof(float)) );
	gpuErrChk( hipMalloc(&dev_statCentralMoments4, statArraySize * sizeof(float)) );
	gpuErrChk( hipMalloc(&dev_statCentralMoments5, statArraySize * sizeof(float)) );
	
	// device malloc: skewness and kurtosis
	gpuErrChk( hipMalloc(&dev_statSkewness, statArraySize * sizeof(float)) );
	gpuErrChk( hipMalloc(&dev_statKurtosis, statArraySize * sizeof(float)) );
	
	// copy old histogram to new
	//TODO: try hipMemcpyHostToHost! --> seg fault
	gpuErrChk( hipMemcpy(dev_hist2stat, host_hist2, size_hist2, hipMemcpyHostToDevice) );
	
	// init: mean median max min
	gpuErrChk( hipMemset(dev_statMean, 0, statArraySize * sizeof(float)) );
	gpuErrChk( hipMemset(dev_statMedian, 0, statArraySize * sizeof(unsigned int)) );
	gpuErrChk( hipMemset(dev_statMax, 0, statArraySize * sizeof(unsigned int)) );
	gpuErrChk( hipMemset(dev_statMin, 255, statArraySize * sizeof(unsigned int)) );
	
	// init: central moments
	gpuErrChk( hipMemset(dev_statCentralMoments2, 0.0, statArraySize * sizeof(float)) );
	gpuErrChk( hipMemset(dev_statCentralMoments3, 0.0, statArraySize * sizeof(float)) );
	gpuErrChk( hipMemset(dev_statCentralMoments4, 0.0, statArraySize * sizeof(float)) );
	gpuErrChk( hipMemset(dev_statCentralMoments5, 0.0, statArraySize * sizeof(float)) );
	
	// init: skewness and kurtosis
	gpuErrChk( hipMemset(dev_statSkewness, 0.0, statArraySize * sizeof(float)) );
	gpuErrChk( hipMemset(dev_statKurtosis, 0.0, statArraySize * sizeof(float)) );
	
	cudaPrintfInit();
	
	// kernel call: mean median max min. 31x31 blocks, 256 threads/block
	kernCalcMeanMedianMaxMin<<<blocksPerGrid, dev_hist2_pitch>>>(dev_hist2stat, (imgBlockSizeX*imgBlockSizeY), dev_hist2_pitch, dev_statMean, dev_statMedian, dev_statMax, dev_statMin);
	
	// kernel call: central moments. 31x31 blocks, 256 threads/block
	kernCalcCentralMoments<<<blocksPerGrid, dev_hist2_pitch>>>(dev_hist2stat, dev_hist2_pitch, (imgBlockSizeX*imgBlockSizeY), dev_statMean, dev_statCentralMoments2, dev_statCentralMoments3, dev_statCentralMoments4, dev_statCentralMoments5);
	
	// kernel call: skewness and kurtosis. 31x31 blocks, 1 thread per block
	kernCalcSkewnessKurtosis<<<blocksPerGrid, 1>>>(dev_statCentralMoments2, dev_statCentralMoments3, dev_statCentralMoments4, dev_statSkewness, dev_statKurtosis);
	
	cudaPrintfDisplay(stdout, true);
	cudaPrintfEnd();
	
	gpuErrChk( hipPeekAtLastError() );
	gpuErrChk( hipDeviceSynchronize() );
	
	// dev to host: mean median max min
	gpuErrChk( hipMemcpy(host_statMean, dev_statMean, statArraySize * sizeof(float), hipMemcpyDeviceToHost) );
	gpuErrChk( hipMemcpy(host_statMedian, dev_statMedian, statArraySize * sizeof(unsigned int), hipMemcpyDeviceToHost) );
	gpuErrChk( hipMemcpy(host_statMax, dev_statMax, statArraySize * sizeof(unsigned int), hipMemcpyDeviceToHost) );
	gpuErrChk( hipMemcpy(host_statMin, dev_statMin, statArraySize * sizeof(unsigned int), hipMemcpyDeviceToHost) );
	
	// dev to host: central moments
	gpuErrChk( hipMemcpy(host_statCentralMoment2, dev_statCentralMoments2, statArraySize * sizeof(float), hipMemcpyDeviceToHost) );
	gpuErrChk( hipMemcpy(host_statCentralMoment3, dev_statCentralMoments3, statArraySize * sizeof(float), hipMemcpyDeviceToHost) );
	gpuErrChk( hipMemcpy(host_statCentralMoment4, dev_statCentralMoments4, statArraySize * sizeof(float), hipMemcpyDeviceToHost) );
	gpuErrChk( hipMemcpy(host_statCentralMoment5, dev_statCentralMoments5, statArraySize * sizeof(float), hipMemcpyDeviceToHost) );
	
	// dev to host: skewness and kurtosis
	gpuErrChk( hipMemcpy(host_statSkewness, dev_statSkewness, statArraySize * sizeof(float), hipMemcpyDeviceToHost) );
	gpuErrChk( hipMemcpy(host_statKurtosis, dev_statKurtosis, statArraySize * sizeof(float), hipMemcpyDeviceToHost) );
	
	// timer stop
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time_gpuStatCalc, start, stop);
	
	printf("\nGPU statistical calculation took %.2f ms, %.2fx %s than CPU\n", time_gpuStatCalc, time_cpuStatCalc/time_gpuStatCalc, (time_gpuStatCalc<time_cpuStatCalc)?"faster":"slower");
	printf("blocks per grid = (%d, %d)\n", blocksPerGrid.x, blocksPerGrid.y);
	printf("threads per block = (%d, %d)\n\n", threadsPerBlock.x, threadsPerBlock.y);

	printf(">>> SAMPLE for block (%d,%d)\n", tmp_whichBlockX, tmp_whichBlockY);
	printf("mean = %f\n", host_statMean[statArrayPitch * tmp_whichBlockY + tmp_whichBlockX]);
	printf("median = %d\n", host_statMedian[statArrayPitch * tmp_whichBlockY + tmp_whichBlockX]);
	printf("max = %d\n", host_statMax[statArrayPitch * tmp_whichBlockY + tmp_whichBlockX]);
	printf("min = %d\n", host_statMin[statArrayPitch * tmp_whichBlockY + tmp_whichBlockX]);
	
	printf("\nCentral Moments\n");
	printf("M2 = %.3f\n", host_statCentralMoment2[statArrayPitch * tmp_whichBlockY + tmp_whichBlockX]);
	printf("M3 = %.3f\n", host_statCentralMoment3[statArrayPitch * tmp_whichBlockY + tmp_whichBlockX]);
	printf("M4 = %.3f\n", host_statCentralMoment4[statArrayPitch * tmp_whichBlockY + tmp_whichBlockX]);
	printf("M5 = %.3f\n", host_statCentralMoment5[statArrayPitch * tmp_whichBlockY + tmp_whichBlockX]);
	
	printf("\n");
	printf("variance = %.3f\n", host_statCentralMoment2[statArrayPitch * tmp_whichBlockY + tmp_whichBlockX]);
	printf("skewness = %.3f\n", host_statSkewness[statArrayPitch * tmp_whichBlockY + tmp_whichBlockX]);
	printf("kurtosis = %.3f\n", host_statKurtosis[statArrayPitch * tmp_whichBlockY + tmp_whichBlockX]);

	/**
	// testing cuprintf
	printf("testing cuprintf\n");
	cudaPrintfInit();
	//kernCalcBlockHist<<<dim3(2,2), dim3(2,2)>>>();
	cudaPrintfDisplay(stdout, true);
	cudaPrintfEnd();
	*/

	// cleanup
	gpuErrChk( hipFree(dev_image) );
	gpuErrChk( hipFree(dev_hist2) );
	
	// gpu histogram 
	
	gpuErrChk( hipFree(dev_hist2stat) );
	gpuErrChk( hipFree(dev_statMean) );
	gpuErrChk( hipFree(dev_statMedian) );
	gpuErrChk( hipFree(dev_statMax) );
	gpuErrChk( hipFree(dev_statMin) );
	
	gpuErrChk( hipDeviceReset() );
	return 0;
}
