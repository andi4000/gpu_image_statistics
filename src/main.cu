#include "hip/hip_runtime.h"
/**
 * //TODO:
 * - need to split this into cpp, h, cu
 * - cpp and h compiled with g++, and nvcc for cu files
 * - how to separate compilation in cmake and then later link?
 * 
 */
#include <cstdio>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>

#include "kernCalcCornerBlockHist.cu"
#include "kernCalcBlockHist.cu"

using namespace std;
using namespace cv;

void processHistogram(unsigned int * hist, int max, bool show=false){
	float mean = 0;
	float sum = 0;
	int n = 0;
	
	for (int i = 0; i < max; i++){
		if (show)
			printf("%d\t%d\n", i, hist[i]);
		sum += i * hist[i];
		n += hist[i];
	}
	mean = sum / (float)n;
	printf("sum = %f\n", sum);
	printf("n = %d\n", n);
	printf("mean = %.5f\n", mean);
}

int main (int argc, char** argv){
	Mat matSrc;
	
	if (argc == 2){
		matSrc = imread(argv[1], 0);
	} else {
		printf("Usage: %s [image file]!\n", argv[0]);
		return -1;
	}
	
	// block sizes
	int imgBlockSizeX = 32, imgBlockSizeY = 32;
	int beginX = 0, beginY = 0;
	
	// image data
	unsigned char * host_image = matSrc.data;
	unsigned int host_hist[256] = {0};
	
	// device image
	unsigned char * dev_image;
	size_t size = matSrc.rows * matSrc.cols * sizeof(unsigned char);
	hipMalloc(&dev_image, size);
	hipMemcpy(dev_image, host_image, size, hipMemcpyHostToDevice);
	
	// device histogram
	unsigned int * dev_hist;
	hipMalloc(&dev_hist, 256 * sizeof(unsigned int));
	hipMemset(dev_hist, 0, 256 * sizeof(unsigned int));
	
	// cuda grid and thread
	dim3 blocksPerGrid = dim3(1,1,1);
	dim3 threadsPerBlock = dim3(imgBlockSizeX, imgBlockSizeY, 1);
	
	// cuda timers
	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	// corner histogram
	hipEventRecord(start, 0);
	kernCalcCornerBlockHist<<<blocksPerGrid, threadsPerBlock>>>(dev_image, matSrc.rows, matSrc.cols, beginX, beginY, dev_hist);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("GPU Histogram took %.5f ms\n", elapsedTime);
	
	// processing result
	hipMemcpy(host_hist, dev_hist, 256 * sizeof(unsigned int), hipMemcpyDeviceToHost);
	printf("Histogram from GPU\n");
	processHistogram(host_hist, 256);
	
	
	// =================== real deal =============================
	
	// grid and thread
	int strideX = 16, strideY = 16;
	
	int gpuBlockTotalX = matSrc.cols / strideX;
	int gpuBlockTotalY = matSrc.rows / strideY;
	blocksPerGrid = dim3(gpuBlockTotalX-1, gpuBlockTotalY-1, 1);
	threadsPerBlock = dim3(imgBlockSizeX, imgBlockSizeY, 1);
	
	// host 2d histogram
	//unsigned int host_hist3[gpuBlockTotalX][gpuBlockTotalY][256] = {{{0}}};
	
	// device 2d histogram
	unsigned int *** dev_hist3;
	size_t size_hist3 = gpuBlockTotalX * gpuBlockTotalY * 256 * sizeof(unsigned int);
	hipMalloc(&dev_hist3, size_hist3);
	hipMemset(dev_hist3, 0, size_hist3);
	
	// main show
	printf("Running the real deal\n");
	printf("blocks per grid = (%d, %d)\n", gpuBlockTotalX-1, gpuBlockTotalY-1);
	printf("threads per block = (%d, %d)\n", imgBlockSizeX, imgBlockSizeY);
	
	cudaPrintfInit();
	kernCalcBlockHist<<<blocksPerGrid, threadsPerBlock>>>(dev_image, matSrc.rows, matSrc.cols, strideX, strideY, dev_hist3);
	cudaPrintfDisplay(stdout, true);
	cudaPrintfEnd();
	
	// result processing
	//hipMemcpy(host_hist3, dev_hist3, size_hist3, hipMemcpyDeviceToHost);
	
	/**
	// testing cuprintf
	printf("testing cuprintf\n");
	cudaPrintfInit();
	kernCalcBlockHist<<<dim3(2,2), dim3(2,2)>>>();
	cudaPrintfDisplay(stdout, true);
	cudaPrintfEnd();
	*/
	
	// cleanup
	hipFree(dev_image);
	hipFree(dev_hist);
	hipDeviceReset();
	return 0;
}
