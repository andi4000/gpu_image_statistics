#include "hip/hip_runtime.h"
#include "cuPrintf.cu"

// kernel with 3D histogram, doesnt work
__global__
void kernCalcBlockHist(
	unsigned char * src,
	int rows,
	int cols,
	int strideX,
	int strideY,
	unsigned int *** outHist
)
{
	int tid_x = blockIdx.x * strideX + threadIdx.x;
	int tid_y = cols * blockIdx.y * strideY + cols * threadIdx.y;
	int tid = tid_y + tid_x;
	
	//TODO: this gives 0s
	atomicAdd(&(outHist[blockIdx.x][blockIdx.y][src[tid]]), 1);
	/**
	// testing. change block idx values to check if tid correct.
	if (blockIdx.x == 30 && blockIdx.y == 30)
		cuPrintf("\tblock (%d, %d)\tthread (%d, %d)\ttid (%d + %d)\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, tid_y, tid_x);
	*/
}

// kernel with pseudo 2d array histogram.
__global__
void kernCalcBlockHist(
	unsigned char * src,
	int rows,
	int cols,
	int strideX,
	int strideY,
	unsigned int * outHist,
	int hist_pitch
)
{
	int tid_x = blockIdx.x * strideX + threadIdx.x;
	int tid_y = cols * blockIdx.y * strideY + cols * threadIdx.y;
	int tid = tid_y + tid_x;
	
	int hist_id = hist_pitch * ( blockDim.x * blockIdx.y + blockIdx.x ) + src[tid];
	atomicAdd(&(outHist[ hist_id ]), 1);
}

// kernel with histogram only for 1 block. works fine but not practical.
__global__
void kernCalcBlockHist(
	unsigned char * src,
	int rows,
	int cols,
	int strideX,
	int strideY,
	unsigned int * outHist
)
{
	int tid_x = blockIdx.x * strideX + threadIdx.x;
	int tid_y = cols * blockIdx.y * strideY + cols * threadIdx.y;
	int tid = tid_y + tid_x;
	
	if (blockIdx.x == 30 && blockIdx.y == 30)
		atomicAdd(&(outHist[src[tid]]), 1);
}

__device__
void fillHistogram()
{
	
}
